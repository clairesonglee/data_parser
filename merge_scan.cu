#include "hip/hip_runtime.h"
//#include "stdint.h"
#include <cstdint>
#include <iostream>
#include <fstream>
#include <string>
#include <chrono>
#include <hipcub/hipcub.hpp>
#include <stdio.h> 

using namespace std;

#define NUM_STATES  4
#define NUM_CHARS   256
#define NUM_THREADS 256
#define NUM_LINES   30

typedef std::chrono::high_resolution_clock Clock;

__constant__ int     d_D[NUM_STATES * NUM_CHARS];
__constant__ uint8_t d_E[NUM_STATES * NUM_CHARS];

template <int states>
struct __align__(4) state_array{
    uint8_t v[states];

    __device__ state_array() {
        for(int i = 0; i < states; i++)
            v[i] = i;
    }

    __device__ void set_SA(int index, int x) {
	   v[index] = x;
    }

};

struct SA_op {
    __device__ state_array<NUM_STATES> operator()(state_array<NUM_STATES> &a, state_array<NUM_STATES> &b){
        state_array<NUM_STATES> c;
        for(int i = 0; i < NUM_STATES; i ++) 
            c.v[i] = b.v[a.v[i]];
        
        return c;
    }
};

__global__
void merge_scan (int num_chars, char* line, int* len_array, int array_len, int* output_array){

    typedef state_array<NUM_STATES> SA;
    typedef hipcub::BlockScan<state_array<NUM_STATES>, NUM_THREADS> BlockScan;
  //  typedef hipcub::BlockScan<int, NUM_THREADS> BlockScan2;

    __shared__ typename BlockScan::TempStorage temp_storage;
   // __shared__ typename BlockScan2::TempStorage temp_storage2;



    int idx = threadIdx.x;
    int block_num = blockIdx.x;

    int len = len_array[block_num];

    if(idx <= len) {

        SA a = SA();
    	SA b = SA();
    	SA temp = SA();
    	for(int i = 0; i < NUM_STATES; i++){
            char c = line[idx + block_num * array_len];
            int x = d_D[(int)(i* NUM_CHARS + c)];
    	    a.set_SA(i, x);
    	}

        BlockScan(temp_storage).InclusiveScan(a, b, SA_op());

        if(idx > 0) {
            char c = line[idx + block_num * array_len];
            int state = b.v[0];
            output_array[idx + block_num * array_len ] = (int) d_E[(int) (NUM_CHARS * state + c)];
            /*
            int start = (int) d_E[(int) (NUM_CHARS * state + c)];
            int end;
            BlockScan2(temp_storage2).InclusiveSum(start, end);
            output_array[idx - 1] = end;
            */
        }
    }

}

__global__
void clear_array (int* input_array, int len) {

    int idx = threadIdx.x + blockIdx.x * blockDim.x;
    if(idx < len) {
        input_array[idx] = 0;
    }

}


//CPU functions

int     D[NUM_STATES][NUM_CHARS];
uint8_t E[NUM_STATES][NUM_CHARS];

void add_transition (int state, uint8_t input, int next_state) 
{
    D[state][input] = next_state;
}

void add_default_transition(int state, int next_state) 
{
    for (int i = 0; i < NUM_CHARS; i++) 
        D[state][i] = next_state;
}

void add_emission(int state, uint8_t input, uint8_t value) 
{
    E[state][input] = value;
}

void add_default_emission(int state, uint8_t value) 
{
    for (int i = 0; i < NUM_CHARS; i++) 
        E[state][i] = value;
}

void Dtable_generate() 
{
    for (int i = 0; i < NUM_STATES; i++) 
        add_default_transition(i ,i);
    
    add_default_transition(2 , 1);
    add_default_transition(3 , 0);
    add_transition(0, '\\', 3);
    add_transition(0, '[', 1);
    add_transition(1, '\\', 2);
    add_transition(1, ']', 0);
}

void Etable_generate() 
{
    for(int i = 0; i < NUM_STATES; i++) 
        add_default_emission(i, 0);
    
    add_emission(0, ',', 1);
}

int max_length()
{
    std::ifstream is("./input_file.txt");   // open file
    string line;
    int length = 0; 

    while (getline(is, line)){
        if(length < line.length())
            length = line.length();
    }
    is.close();
    
    return length; 
}


int main() {

    Dtable_generate();
    Etable_generate();
    const int array_len = max_length(); 

    hipMemcpyToSymbol(HIP_SYMBOL(d_D), D, NUM_STATES * NUM_CHARS * sizeof(int));
    hipMemcpyToSymbol(HIP_SYMBOL(d_E), E, NUM_STATES * NUM_CHARS * sizeof(uint8_t));

    int* h_output_array = new int[NUM_LINES * array_len];

    //Memory allocation for kernel functions
    
    int* d_output_array;
    hipMalloc((int**)&d_output_array, array_len * sizeof(int) * NUM_LINES);

    char* d_line;
    hipMalloc((char**) &d_line, array_len * sizeof(char) * NUM_LINES);

    int* d_len_array;
    hipMalloc((char**) &d_len_array, NUM_LINES * sizeof(int));


    std::ifstream is("./input_file.txt");

    string line;
    char* input_strings = new char[NUM_LINES * array_len];

    int len_array[NUM_LINES];
    int count = 0;

    //start timer
    auto t1 = Clock::now();
    while (getline(is, line)) 
    { 

        for(int i = 0; i < array_len; i++) {
            if(i < line.length())
                input_strings[count * array_len + i] = line[i];
            else
                input_strings[count * array_len + i] = 0;
        }

        len_array[count] = line.length();
        count++;

        if(count == NUM_LINES){
          
           // len = line.length();
            hipMemcpy(d_line, input_strings, array_len * sizeof(char) * NUM_LINES, hipMemcpyHostToDevice);     
            hipMemcpy(d_len_array, len_array, NUM_LINES * sizeof(int), hipMemcpyHostToDevice);     


            dim3 dimGrid(NUM_LINES,1,1);
            dim3 dimBlock(NUM_THREADS,1,1);
            merge_scan<<<dimGrid, dimBlock>>>(1, d_line, d_len_array, array_len, d_output_array);
           
            hipMemcpy(h_output_array, d_output_array, array_len  * sizeof(int) * NUM_LINES, hipMemcpyDeviceToHost);
            
            for(int j = 0; j < NUM_LINES; j++) {
                for(int i = 0; i < array_len; i++) {
                   if(h_output_array[i + j * array_len] == 1) 
                       cout << i << " "; 
                }
                cout << endl;
            }
    
            clear_array<<<dimGrid, dimBlock>>>(d_output_array, array_len * NUM_LINES);

            count = 0;
        }
    }

    //if the total number of lines is not a multiple of NUM_LINES
    if(count != 0) {

        hipMemcpy(d_line, input_strings, array_len * sizeof(char) * NUM_LINES, hipMemcpyHostToDevice);     
        hipMemcpy(d_len_array, len_array, NUM_LINES * sizeof(int), hipMemcpyHostToDevice);     


        dim3 dimGrid(count,1,1);
        dim3 dimBlock(NUM_THREADS,1,1);
        merge_scan<<<dimGrid, dimBlock>>>(1, d_line, d_len_array, array_len, d_output_array);
       
        hipMemcpy(h_output_array, d_output_array, array_len  * sizeof(int) * NUM_LINES, hipMemcpyDeviceToHost);
        
        for(int j = 0; j < count; j++) {
            for(int i = 0; i < array_len; i++) {
               if(h_output_array[i + j * array_len] == 1) 
                   cout << i << " "; 
            }
            cout << endl;
        }

    }

    //end timer
    is.close();

    auto t2 = Clock::now();
    cout << std::chrono::duration_cast<std::chrono::microseconds>(t2 - t1).count() << " microseconds" << endl;
    
    hipFree(d_output_array);
    hipFree(d_line);

    free(h_output_array);
    free(input_strings);


    return 0;
}

