#include "hip/hip_runtime.h"

#include <cstdint>
#include <iostream>
#include <fstream>
#include <string>
#include <chrono>
#include <hipcub/hipcub.hpp>

#include <stdio.h> 

using namespace std;

#define NUM_STATES 4
#define NUM_CHARS  256
#define NUM_THREADS 256
#define NUM_LINES 322
#define NUM_BLOCKS 30

#define BUFFER_SIZE 250000
#define NUM_COMMAS 50
#define INPUT_FILE "./input_file.txt"

typedef std::chrono::high_resolution_clock Clock;

//Transition table for GPU function
__constant__ int     d_D[NUM_STATES * NUM_CHARS];
//Emission table for GPU function
__constant__ uint8_t d_E[NUM_STATES * NUM_CHARS];




template <int states>
struct __align__(4) state_array{
    uint8_t v[states];

    __device__ state_array() {
        for(int i = 0; i < states; i++)
            v[i] = i;
    }

    __device__ void set_SA(int index, int x) {
       v[index] = x;
    }

};

typedef state_array<NUM_STATES> SA;

//a = b
__device__ void SA_copy(SA &a, SA &b) {
    for(int i = 0; i < NUM_STATES; i ++) 
        a.v[i] = b.v[i];
}

struct SA_op {
    __device__ SA operator()(SA &a, SA &b){
        SA c;
        for(int i = 0; i < NUM_STATES; i++) 
            c.v[i] = b.v[a.v[i]];
        
        return c;
    }
};

__global__
void remove_empty_elements (int* input, int* len_array, int total_lines, int* index, int* temp_base, 
                            int* offset_array,  int* output) {

    __shared__ int line_num;
    __shared__ int base;

    int len;
    int block_num;


    if(threadIdx.x == 0) 
        line_num = atomicInc((unsigned int*) &index[0], INT_MAX);
    __syncthreads();
    block_num =  line_num;

    

    while(block_num < total_lines) {

        len = len_array[block_num];


        if(threadIdx.x == 0)
            base = atomicAdd(&temp_base[0], len);
        __syncthreads();
        

        offset_array[block_num] = (base);

        for(int loop = threadIdx.x; loop < len; loop += NUM_THREADS) {

            if(loop < len)
                output[base + loop] = input[block_num * NUM_COMMAS + loop];
        }

        if(threadIdx.x == 0) 
            line_num = atomicInc((unsigned int*) &index[0], INT_MAX);
         __syncthreads();
        block_num =  line_num;



    }




}



__global__
void merge_scan (char* line, int* len_array, int* offset_array, int* output_array, 
                 int* index, int total_lines, int* num_commas_array){


    typedef hipcub::BlockScan<SA, NUM_THREADS + 1> BlockScan; // change name
    typedef hipcub::BlockScan<int, NUM_THREADS + 1> BlockScan2; //

    __shared__ typename BlockScan::TempStorage temp_storage;
    __shared__ typename BlockScan2::TempStorage temp_storage2;
    __shared__ SA prev_value;
    __shared__ int prev_sum;
    __shared__ int line_num;

    int len, offset;
    int block_num;

    if(threadIdx.x == 0) {
        line_num = atomicInc((unsigned int*) &index[0], INT_MAX);
    //   printf("block_num: %d\n", line_num);
    }
    __syncthreads();
    block_num =  line_num;

    while(block_num < total_lines ) {

        len = len_array[block_num];
        offset = offset_array[block_num];

        //initialize starting values
        SA a = SA();
        SA_copy(prev_value , a);

        prev_sum = 0;
        int loop;

        //If the string is longer than NUM_THREADS
        for(int ph = 0; ph < (int)(len / (int)NUM_THREADS) + 1; ph++) {

            loop = threadIdx.x + ((int)NUM_THREADS) * ph;
            char c = 0;
            //__syncthreads();
            if(loop < len) {
                c = line[loop + offset];

                //Check that it has to fetch the data from the previous loop
                if(loop % NUM_THREADS == 0) {
                    SA_copy(a, prev_value);
                }

                else {   
                    for(int i = 0; i < NUM_STATES; i++){
                        int x = d_D[(int)(i* NUM_CHARS + c)];
                        a.set_SA(i, x);
                    }
                }
            }
            __syncthreads();

            BlockScan(temp_storage).InclusiveScan(a, a, SA_op());
            __syncthreads();

            int state = a.v[0];
            int start = (int) d_E[(int) (NUM_CHARS * state + c)];
            int end;
            BlockScan2(temp_storage2).InclusiveSum(start, end);
            if(start == 1 && loop < len) {
                output_array[end - 1 + block_num * NUM_COMMAS + prev_sum] = loop;
            }

            //save the values for the next loop
            if((loop + 1) % NUM_THREADS == 0) {
                SA_copy(prev_value , a);
                prev_sum += end;
             //   printf("loop: %d, block_num: %d, blcok_ID: %d, prev_sum: %d\n", loop, block_num, blockIdx.x, prev_sum);
            }
            __syncthreads();
                    //save the number of commas in the current line
        }

        if(loop == len - 1) 
            num_commas_array[block_num] = prev_sum;

        if(threadIdx.x == 0) 
            line_num = atomicInc((unsigned int*) &index[0], INT_MAX);
         __syncthreads();
        block_num =  line_num;
    }


}

__global__
void clear_array (int* input_array, int len) {

    int idx = threadIdx.x + blockIdx.x * blockDim.x;
    if(idx < len) {
        input_array[idx] = 0;
    }

}


//CPU functions

int     D[NUM_STATES][NUM_CHARS];
uint8_t E[NUM_STATES][NUM_CHARS];

void add_transition (int state, uint8_t input, int next_state) 
{
    D[state][input] = next_state;
}

void add_default_transition(int state, int next_state) 
{
    for (int i = 0; i < NUM_CHARS; i++) 
        D[state][i] = next_state;
}

void add_emission(int state, uint8_t input, uint8_t value) 
{
    E[state][input] = value;
}

void add_default_emission(int state, uint8_t value) 
{
    for (int i = 0; i < NUM_CHARS; i++) 
        E[state][i] = value;
}

void Dtable_generate() 
{
    for (int i = 0; i < NUM_STATES; i++) 
        add_default_transition(i ,i);
    
    add_default_transition(2 , 1);
    add_default_transition(3 , 0);

    add_transition(0, '[', 1);
    add_transition(1, '\\', 2);
    add_transition(1, ']', 0);
    add_transition(0, '\\', 3);
}

void Etable_generate() 
{
    for(int i = 0; i < NUM_STATES; i++) 
        add_default_emission(i, 0);
    
    add_emission(0, ',', 1);
}

int max_length()
{
    std::ifstream is(INPUT_FILE);   // open file
    string line;
    int length = 0; 

    while (getline(is, line)){
        if(length < line.length())
            length = line.length();
    }
    is.close();
    
    return length; 
}



int main() {

    Dtable_generate();
    Etable_generate();

    hipMemcpyToSymbol(HIP_SYMBOL(d_D), D, NUM_STATES * NUM_CHARS * sizeof(int));
    hipMemcpyToSymbol(HIP_SYMBOL(d_E), E, NUM_STATES * NUM_CHARS * sizeof(uint8_t));

    int* h_output_array = new int[BUFFER_SIZE];

    std::ifstream is(INPUT_FILE);

    // get length of file:
    is.seekg (0, std::ios::end);
    long length = is.tellg();
    is.seekg (0, std::ios::beg);

    if(length > BUFFER_SIZE){
        cout<<"Error: File is too large to be read to buffer"<<endl;
    }
    else{
        string line; 
        long line_length;
        long line_count = 0; 
        long char_offset = 0; 

        // allocate memory:
        char* buffer = new char [BUFFER_SIZE];
        int* len_array = new int[NUM_LINES];
        int* offset_array = new int[NUM_LINES];
        int* comma_offset_array = new int[NUM_LINES];
        int* comma_len_array = new int [NUM_LINES];

        offset_array[0] = 0;

        while (getline(is, line)){

            line_length = line.size();

            // keep track of lengths of each line
            len_array[line_count] = line_length;

            // update offset from start of file
            char_offset += line_length + 1;
            offset_array[line_count + 1] = char_offset;

            // increment line index
            line_count++;

        }
        is.close();
        // reopen file stream
        std::ifstream is(INPUT_FILE);

        // read data as a block:
        is.read (buffer,length);
        //cout<<"buffer "<<buffer<<endl;

       // int* h_num_commas = new int[line_count];

        //Memory allocation for kernel functions
    


        int* d_output_array;
        hipMalloc((int**)&d_output_array, line_count * NUM_COMMAS * sizeof(int));

        char* d_buffer;
        hipMalloc((char**) &d_buffer, BUFFER_SIZE * sizeof(char));

        int* d_len_array;
        hipMalloc((int**) &d_len_array, line_count * sizeof(int));

        int* d_offset_array;
        hipMalloc((int**) &d_offset_array, line_count * sizeof(int));

        int* d_num_commas;
        hipMalloc((int**) &d_num_commas, line_count * sizeof(int));

        int* d_final_array;
        hipMalloc((int**) &d_final_array, BUFFER_SIZE * sizeof(int));

        int* d_comma_offset_array;
        hipMalloc((int**) &d_comma_offset_array, line_count * sizeof(int));


        int* d_stack;
        hipMalloc((int**) &d_stack, sizeof(int));

        int* d_temp_base;
        hipMalloc((int**) &d_temp_base, sizeof(int));





        int temp = 0;

        auto t1 = Clock::now();

        hipMemcpy(d_buffer, buffer, BUFFER_SIZE * sizeof(char), hipMemcpyHostToDevice);     
        hipMemcpy(d_len_array, len_array, line_count * sizeof(int), hipMemcpyHostToDevice);     
        hipMemcpy(d_offset_array, offset_array, line_count * sizeof(int), hipMemcpyHostToDevice);    
        hipMemcpy(d_stack, &temp, sizeof(int), hipMemcpyHostToDevice);
        hipMemcpy(d_temp_base, &temp, sizeof(int), hipMemcpyHostToDevice);


        auto t2 = Clock::now();

        cout <<"Host to Device:" <<std::chrono::duration_cast<std::chrono::microseconds>(t2 - t1).count() << " microseconds" << endl;

        dim3 dimGrid(NUM_BLOCKS,1,1);
        dim3 dimBlock(NUM_THREADS,1,1);

        auto t3 = Clock::now();

       merge_scan<<<dimGrid, dimBlock>>>(d_buffer, d_len_array, d_offset_array, d_output_array, d_stack, line_count, d_num_commas);

        hipDeviceSynchronize();

        hipMemcpy(d_stack, &temp, sizeof(int), hipMemcpyHostToDevice);

        hipDeviceSynchronize();

        remove_empty_elements<<<dimGrid, dimBlock>>> (d_output_array, d_num_commas, line_count, d_stack, d_temp_base, d_comma_offset_array, d_final_array);

        hipDeviceSynchronize();

        auto t4 = Clock::now();
        cout << "data trans:" << std::chrono::duration_cast<std::chrono::microseconds>(t4 - t3).count() << " microseconds" << endl;


        auto t5 = Clock::now();
        //change the size later
        hipMemcpy(h_output_array, d_final_array, line_count * NUM_COMMAS * sizeof(int), hipMemcpyDeviceToHost);
        hipMemcpy(comma_len_array, d_num_commas, line_count * sizeof(int), hipMemcpyDeviceToHost);
        hipMemcpy(comma_offset_array, d_comma_offset_array, line_count * sizeof(int), hipMemcpyDeviceToHost);
        auto t6 = Clock::now();
        cout << "Device to Host:" << std::chrono::duration_cast<std::chrono::microseconds>(t6 - t5).count() << " microseconds" << endl;

        
        
         for(int i = 0; i < line_count; i++) {
            int len = comma_len_array[i];
           
            int off = comma_offset_array[i];
            for(int j = 0; j < len; j++) {
                cout << h_output_array[off + j] << " ";
            }
             // cout << len;
            cout << endl;
        }
        
        

        

        //clear_array<<<dimGrid, dimBlock>>>(d_output_array, BUFFER_SIZE);

        // close filestream
        is.close();


        hipFree(d_output_array);
        hipFree(d_buffer);
        hipFree(d_len_array);
        hipFree(d_offset_array);
        hipFree(d_comma_offset_array);
        hipFree(d_stack);
        hipFree(d_temp_base);
        hipFree(d_num_commas);

        // delete temporary buffers
        delete [] buffer;
        delete [] len_array;
        delete [] offset_array;
        delete [] comma_offset_array;
        delete [] comma_len_array;

    }
    delete [] h_output_array;



    return 0;
}


